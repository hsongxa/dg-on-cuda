/**
 * MIT License
 * 
 * Copyright (c) 2021 hsongxa
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 **/

#include "k_stokes_2d.cuh"

#include <cassert>

#include "device_SemiDiscOp_wrapper.cuh"
#include "explicit_runge_kutta.h"
#include "axpy.h"

// NOTE: The sole purpose of this .cu file is to have an entry point to start the nvcc compilation
// NOTE: as all the rest code is in header files only (except the main() function). The reason we
// NOTE: do not instantiate kernel templates in the main() funciton is that .cpp files are compiled
// NOTE: by "g++ -std=c++17" whereas the .cu files are compiled by "nvcc -std=c++14" due to the CUDA
// NOTE: version we use. If we could use c++17 for CUDA code, we wouldn't need this .cu file -- we
// NOTE: could simply instantiate the kernel templates in the main() function and change main.cpp
// NOTE: to main.cu.

__constant__ double Dr[MAX_NUM_CELL_NODES * MAX_NUM_CELL_NODES];
__constant__ double Ds[MAX_NUM_CELL_NODES * MAX_NUM_CELL_NODES];
__constant__ double L[MAX_NUM_CELL_NODES * 3 * MAX_NUM_FACE_NODES];

d_stokes_2d<double, int>* create_device_object(int num_cells, int order,
                                               const DIntVector& face_0_nodes,
                                               const DIntVector& face_1_nodes,
                                               const DIntVector& face_2_nodes,
                                               const DDblVector& inv_jacobians,
                                               const DDblVector& Js,
                                               const DDblVector& face_Js,
                                               const DIntVector& interface_cells,
                                               const DIntVector& interface_faces,
                                               int num_boundary_nodes,
                                               const DDblVector& boundary_node_Xs,
                                               const DDblVector& boundary_node_Ys,
                                               const DDblVector& outward_normal_Xs,
                                               const DDblVector& outward_normal_Ys)
{
  assert(num_cells > 0);
  assert(order > 0 && order < 7);

//  hipMemcpyToSymbol(HIP_SYMBOL(Dr), dr, (order + 1) * (order + 1) * (order + 2) * (order + 2) / 4 * sizeof(double));
//  hipMemcpyToSymbol(HIP_SYMBOL(Ds), ds, (order + 1) * (order + 1) * (order + 2) * (order + 2) / 4 * sizeof(double));
//  hipMemcpyToSymbol(HIP_SYMBOL(L), l, (order + 1) * (order + 2) * 3 * (order + 1) / 2 * sizeof(double));
//
//  double *d_Dr, *d_Ds, *d_L;
//  hipGetSymbolAddress((void**)&d_Dr, Dr);
//  hipGetSymbolAddress((void**)&d_Ds, Ds);
//  hipGetSymbolAddress((void**)&d_L, L);
  
  d_stokes_2d<double, int> tmp;
  tmp.NumCells = num_cells;
  tmp.Order = order;
  tmp.Face_0_Nodes = thrust::raw_pointer_cast(face_0_nodes.data());
  tmp.Face_1_Nodes = thrust::raw_pointer_cast(face_1_nodes.data());
  tmp.Face_2_Nodes = thrust::raw_pointer_cast(face_2_nodes.data());
  tmp.Inv_Jacobian = thrust::raw_pointer_cast(inv_jacobians.data());
  tmp.J = thrust::raw_pointer_cast(Js.data());
  tmp.Face_J = thrust::raw_pointer_cast(face_Js.data());
  tmp.Interfaces_Cell = thrust::raw_pointer_cast(interface_cells.data());
  tmp.Interfaces_Face = thrust::raw_pointer_cast(interface_faces.data());
  tmp.Boundary_Nodes_X = thrust::raw_pointer_cast(boundary_node_Xs.data());
  tmp.Boundary_Nodes_Y = thrust::raw_pointer_cast(boundary_node_Ys.data());
  tmp.Outward_Normals_X = thrust::raw_pointer_cast(outward_normal_Xs.data());
  tmp.Outward_Normals_Y = thrust::raw_pointer_cast(outward_normal_Ys.data());

  d_stokes_2d<double, int>* dOp;
  hipMalloc((void**)&dOp, sizeof(d_stokes_2d<double, int>));
  hipMemcpy(dOp, &tmp, sizeof(d_stokes_2d<double, int>), hipMemcpyHostToDevice);

  return dOp;
}

void run_on_device(int gridSize, int blockSize, DZipIterator inout, std::size_t size, double t, double dt,
                   d_stokes_2d<double, int>* d_op, DZipIterator wk0, DZipIterator wk1, DZipIterator wk2,
                   DZipIterator wk3, DZipIterator wk4)
{ 
  // NOTE: For the same reason as documented at the beginning of this file, the instantiation of the wrapper object
  // NOTE: has to be here, rather than in the main(). But ideally it should be pulled to the main() and just do the
  // NOTE: instantiation once outside the time advancing loop, instead of repeatedly doing it here at every time step.
  dgc::device_SemiDiscOp_wrapper<d_stokes_2d<double, int>> w;
  w.m_Dop = d_op;
  w.m_GridSize = gridSize;
  w.m_BlockSize = blockSize;

  // TODO: trigger the kernels here
}

void destroy_device_object(d_stokes_2d<double, int>* device_obj)
{
  hipFree(device_obj);
}

