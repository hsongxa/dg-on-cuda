/**
 * MIT License
 * 
 * Copyright (c) 2021 hsongxa
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 **/

#include "k_maxwell_2d.cuh"

#include <cassert>

#include "device_SemiDiscOp_wrapper.cuh"
#include "explicit_runge_kutta.h"
#include "k_axpy.cuh"

// NOTE: The sole purpose of this .cu file is to have an entry point to start the nvcc compilation
// NOTE: as all the rest code is in header files only (except the main() function). The reason we
// NOTE: do not instantiate kernel templates in the main() funciton is that .cpp files are compiled
// NOTE: by "g++ -std=c++17" whereas the .cu files are compiled by "nvcc -std=c++14" due to the CUDA
// NOTE: version we use. If we could use c++17 for CUDA code, we wouldn't need this .cu file -- we
// NOTE: could simply instantiate the kernel templates in the main() function and change main.cpp
// NOTE: to main.cu.

__constant__ double Dr[MAX_NUM_CELL_NODES * MAX_NUM_CELL_NODES];
__constant__ double Ds[MAX_NUM_CELL_NODES * MAX_NUM_CELL_NODES];
__constant__ double L[MAX_NUM_CELL_NODES * 3 * MAX_NUM_FACE_NODES];

d_maxwell_2d<double, int>* create_device_object(int num_cells, int order, double* dr, double* ds, double* l,
                                                int* face_0_nodes, int* face_1_nodes, int* face_2_nodes,
                                                double* inv_jacobians, double* Js, double* face_Js,
                                                int* interface_cells, int* interface_faces, int num_boundary_nodes,
                                                double* boundary_node_Xs, double* boundary_node_Ys,
                                                double* outward_normal_Xs, double* outward_normal_Ys,
                                                int** d_face0_nodes, int** d_face1_nodes, int** d_face2_nodes,
                                                double** d_inv_jacobians, double** d_Js, double** d_face_Js,
                                                int** d_interface_cells, int** d_interface_faces,
                                                double** d_boundary_node_Xs, double** d_boundary_node_Ys,
                                                double** d_outward_normal_Xs, double** d_outward_normal_Ys)
{
  assert(num_cells > 0);
  assert(order > 0 && order < 7);

  hipMemcpyToSymbol(HIP_SYMBOL(Dr), dr, (order + 1) * (order + 1) * (order + 2) * (order + 2) / 4 * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(Ds), ds, (order + 1) * (order + 1) * (order + 2) * (order + 2) / 4 * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(L), l, (order + 1) * (order + 2) * 3 * (order + 1) / 2 * sizeof(double));

  double *d_Dr, *d_Ds, *d_L;
  hipGetSymbolAddress((void**)&d_Dr, Dr);
  hipGetSymbolAddress((void**)&d_Ds, Ds);
  hipGetSymbolAddress((void**)&d_L, L);
  
  dgc::create_simple_discretization_2d_on_device(num_cells, order, face_0_nodes, face_1_nodes, face_2_nodes, inv_jacobians,
                                                 Js, face_Js, interface_cells, interface_faces, num_boundary_nodes,
                                                 boundary_node_Xs, boundary_node_Ys, outward_normal_Xs, outward_normal_Ys,
                                                 d_face0_nodes, d_face1_nodes, d_face2_nodes, d_inv_jacobians, d_Js, d_face_Js,
                                                 d_interface_cells, d_interface_faces, d_boundary_node_Xs, d_boundary_node_Ys,
                                                 d_outward_normal_Xs, d_outward_normal_Ys);


  d_maxwell_2d<double, int> tmp;
  tmp.NumCells = num_cells;
  tmp.Order = order;
  tmp.Dr = d_Dr;
  tmp.Ds = d_Ds;
  tmp.L = d_L;
  tmp.Face_0_Nodes = *d_face0_nodes;
  tmp.Face_1_Nodes = *d_face1_nodes;
  tmp.Face_2_Nodes = *d_face2_nodes;
  tmp.Inv_Jacobian = *d_inv_jacobians;
  tmp.J = *d_Js;
  tmp.Face_J = *d_face_Js;
  tmp.Interfaces_Cell = *d_interface_cells;
  tmp.Interfaces_Face = *d_interface_faces;
  tmp.Boundary_Nodes_X = *d_boundary_node_Xs;
  tmp.Boundary_Nodes_Y = *d_boundary_node_Ys;
  tmp.Outward_Normals_X = *d_outward_normal_Xs;
  tmp.Outward_Normals_Y = *d_outward_normal_Ys;

  d_maxwell_2d<double, int>* dOp;
  hipMalloc((void**)&dOp, sizeof(d_maxwell_2d<double, int>));
  hipMemcpy(dOp, &tmp, sizeof(d_maxwell_2d<double, int>), hipMemcpyHostToDevice);

  return dOp;
}

void rk4_on_device(int gridSize, int blockSize, double* inout, std::size_t size, double t, double dt,
                   d_maxwell_2d<double, int>* d_op, double* wk0, double* wk1, double* wk2, double* wk3, double* wk4)
{ 
  // NOTE: For the same reason as documented at the beginning of this file, the instantiation of the wrapper object
  // NOTE: has to be here, rather than in the main(). But ideally it should be pulled to the main() and just do the
  // NOTE: instantiation once outside the time advancing loop, instead of repeatedly doing it here at every time step.
  dgc::device_SemiDiscOp_wrapper<d_maxwell_2d<double, int>> w;
  w.m_Dop = d_op;
  w.m_GridSize = gridSize;
  w.m_BlockSize = blockSize;

  dgc::rk4(inout, size, t, dt, w, &dgc::k_axpy_auto<double>, wk0, wk1, wk2, wk3, wk4);
}

void destroy_device_object(d_maxwell_2d<double, int>* device_obj, int* d_face0_nodes, int* d_face1_nodes,
                           int* d_face2_nodes, double* d_inv_jacobians, double* d_Js, double* d_face_Js,
                           int* d_interface_cells, int* d_interface_faces, double* d_boundary_node_Xs,
                           double* d_boundary_node_Ys, double* d_outward_normal_Xs, double* d_outward_normal_Ys)
{
  dgc::destroy_simple_discretization_2d_on_device(d_face0_nodes, d_face1_nodes, d_face2_nodes, d_inv_jacobians,
                                                  d_Js, d_face_Js, d_interface_cells, d_interface_faces,
                                                  d_boundary_node_Xs, d_boundary_node_Ys, d_outward_normal_Xs,
                                                  d_outward_normal_Ys);
  hipFree(device_obj);
}

